#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipblas.h>
#include <hiprand.h>


#include "kernels.h"
#include "randgen.h"

// some CPU variables
extern float Htau;
extern int HI;
extern int HK_star;
extern int HK;
extern int V_extended;


const float RHO_UPPER = 2.00f;
const float RHO_LOWER = 0.35f;
const double VOXEL_VOL = 8.0;


// Finds the matrix-vector products
void FIND_EIGEN_PROD (float *rho , float2 *DFT_big , float *dist , hipfftHandle plan_big , float *eigen , float *gamma , float2 *DFT , hipfftHandle plan , float *Cgamma , float *CSgamma) {

    // 2 nested block circulant matrices appear in the calculations. The first kernel finds the bases.
    FindBaseBig<<<K*V/NT,NT>>>(rho,DFT_big,dist);
    // Load the \gamma vector in the small DFT
    LoadVec<<<K*V/NT,NT>>>(gamma,DFT);
    hipDeviceSynchronize();
    // Execute the forward DFT
    hipfftExecC2C(plan_big,DFT_big,DFT_big,HIPFFT_FORWARD);
    hipDeviceSynchronize();
    // Execute the IDFT of the vector
    hipfftExecC2C(plan,DFT,DFT,HIPFFT_BACKWARD);

    // Save the eigenvalues of the correlation matrix. Be careful. These values are multiplied by sqrt(n) already and the square root is taken
    // At the same time prepare the big FFT object for the IDFT(gamma) that is coming. See kernel carefully.
    DragEigenBig<<<K*V/NT,NT>>>(DFT_big,eigen);
    hipDeviceSynchronize();

    // Throw IDFT(\gamma) into the big object
    LoadAddVec<<<K*V/NT,NT>>>(DFT,DFT_big);
    hipDeviceSynchronize();

    // Perform the final DFT
    hipfftExecC2C(plan_big,DFT_big,DFT_big,HIPFFT_FORWARD);
    hipDeviceSynchronize();

    // And finally save the products
    DragRealBig<<<K*V/NT,NT>>>(DFT_big,Cgamma,CSgamma);
    hipDeviceSynchronize();

}


// Finds the design matrix times coefficients product
void FIND_ZB(float *Z , float * B , float * ZB , float *COVARIATES) {

    // Calculate the product
    hipblasSgemv('n',HI,HK_star,1.0f,Z,HI,B,1,0.0f,ZB,1);
    hipDeviceSynchronize();
    // Then save to a vector
    SendToCovariates<<<CUBLAS_TMP,512>>>(ZB,COVARIATES);
    hipDeviceSynchronize();
    
}






// The following function calculates the gradient.
// For grad_gamma the minus sign is take care of when multiplying by sigma
// For grad_beta and grad_sigma the minus sign is taken care of when the partial sums are summed in the CPU
void FIND_GRAD(float *gamma , float *grad_gamma , float *beta , float *grad_beta , float *sigma , float *grad_sigma , float *voxel_tmp , float *ZB , float *Cgamma , float *vol , double *sigma_part , double *beta_part , double *Hsigma_part , double *Hbeta_part , float * COVARIATES , float *TERM_GAMMA , float2 *DFT , float *eigen , hipfftHandle plan , float *Hsigma , float *Hbeta , float *HZN , float *Hrho , float *rho , float *CSgamma , double *rho_part , double *Hrho_part , float *grad_rho) {
    
    // The first kernel calculates common term for all the variables per voxel. It stores it in voxel_tmp.
    GradFirst<<<V/NT,NT>>>(voxel_tmp ,sigma ,Cgamma ,vol , DFT , COVARIATES,TERM_GAMMA);
    hipDeviceSynchronize();

    // The lines below calculate the FFT product that appears in gradient calculations
    // The vector is already loaded in the FFT object so we just find its IDFT
    hipfftExecC2C(plan, DFT, DFT, HIPFFT_BACKWARD);
    hipDeviceSynchronize();

    // Now load the eigenvalues of the square root correlation matrix
    LoadAddVecSecond<<<K*V/NT,NT>>>(eigen,DFT);
    hipDeviceSynchronize();

    // And finally find the matrix vector product
    hipfftExecC2C(plan, DFT, DFT, HIPFFT_FORWARD);
    hipDeviceSynchronize();

    // To finish, send the values to the gradient of \gamma
    DragRealGrad<<<K*V/NT,NT>>>(DFT,grad_gamma,gamma);
    hipDeviceSynchronize();

    // This  series of kernels adds up the partial sums for the gradients of scalar parameters
    GradSigma<<<NB,NT>>>(voxel_tmp , sigma_part , Cgamma , TERM_GAMMA );
    GradBeta<<<NB,NT>>>(voxel_tmp , beta_part);
    GradRho<<<NB,NT>>>(voxel_tmp,rho_part,CSgamma,TERM_GAMMA);
    hipDeviceSynchronize();

    // Move the partial sums and the parameters to the CPU for the adding up
    int s,ss;
    hipMemcpy(Hsigma,sigma, HK*sizeof(float),      hipMemcpyDeviceToHost);
    hipMemcpy(Hrho,  rho,   HK*sizeof(float),      hipMemcpyDeviceToHost);
    hipMemcpy(Hbeta, beta,  HK_star*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Hsigma_part, sigma_part, NB*HK*sizeof(double),      hipMemcpyDeviceToHost);
    hipMemcpy(Hrho_part,   rho_part,   NB*HK*sizeof(double),      hipMemcpyDeviceToHost);
    hipMemcpy(Hbeta_part,  beta_part,  NB*HK_star*sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Dont get confused. I will find the derivatives inside the parameter vectors.
    for (s=0 ; s<HK ; s++) {
        Hrho[s] = Hsigma[s];
        Hsigma[s] *= -Htau;
    }
    for (s=0 ; s<HK_star ; s++) {
        Hbeta[s] *= -Htau;
        Hbeta[s] += HZN[s];
    }

    // First add the partial sums found earlier in the GPU
    double tmp_sigma[HK];
    double tmp_rho[HK];
    double tmp_beta[HK_star];
    for (ss=0 ; ss<NB ; ss++){
        for (s=0 ; s<HK ; s++) {
            tmp_sigma[s] += Hsigma_part[s+HK*ss];
            tmp_rho[s]   += Hrho_part[s+HK*ss];
        }
        for (s=0 ; s<HK_star ; s++) {
            tmp_beta[s] += Hbeta_part[s+HK_star*ss];
        }
    }

    // And then add the sums to the gradients of the parameters
    for (s=0 ; s<HK ; s++) {
            Hsigma[s] += -(float)tmp_sigma[s];
            Hrho[s]   *=  (float)tmp_rho[s]/200; // The 200 appears due to the parametrisation
        }
    for (s=0 ; s<HK_star ; s++) {
            Hbeta[s]  += -(float)tmp_beta[s];
    }


    // And finally send everything back to the GPU
    hipMemcpy(grad_sigma, Hsigma, HK*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(grad_rho,   Hrho,   HK*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(grad_beta,  Hbeta,  HK_star*sizeof(float),hipMemcpyHostToDevice);
    hipDeviceSynchronize();

}







// Finds the log-likelihood
void FIND_LIK(float *lik_tmp , float *gamma , float *COVARIATES , float * sigma , float *Cgamma , float *TERM_GAMMA , float *vol , double *lik_part , double *Hlik_part , float *ZB , float *HZB , float *Hsigma , float *beta , float *Hbeta , double *likelihood , int *Hcounts )
{
    // First do the two kernels required in the GPU
    LikFirst<<<V/NT,NT>>>(lik_tmp,gamma,COVARIATES,sigma,Cgamma,TERM_GAMMA,vol);
    hipDeviceSynchronize();
    LikSecond<<<NB,NT>>>(lik_tmp,lik_part);
    hipDeviceSynchronize();

    // Define two variables that will be used
    int s;
    double sum=0;

    // Transfer everything you need to the CPU
    hipMemcpy(Hlik_part, lik_part, NB*sizeof(double),      hipMemcpyDeviceToHost);
    hipMemcpy(HZB,       ZB,       HI*sizeof(float),      hipMemcpyDeviceToHost);
    hipMemcpy(Hsigma,    sigma,    HK*sizeof(float),      hipMemcpyDeviceToHost);
    hipMemcpy(Hbeta,     beta,     HK_star*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Add the likelihood partial sums together
    for (s=0 ; s<NB ; s++) {
        sum += Hlik_part[s];
    }

    // Add the annoying term that appears in the likelihood equation.
    for (s=0 ; s<HI ; s++) {
        sum += (double) HZB[s]*Hcounts[s];
    }

    // Finally add the prior contributions
    for (s=0 ; s<HK ; s++){
        sum += -0.5*Htau*Hsigma[s]*Hsigma[s];
    }
    for (s=0 ; s<HK_star ; s++){
        sum += -0.5*Htau*Hbeta[s]*Hbeta[s];
    }

    // Now just save the value that you obtained.
    likelihood[0] = sum;

}




// If a move of the HMC is accepted then save the new values
void SAVE_ORIGINALS(double *likelihood , float *beta , float *ori_beta , float *sigma , float *ori_sigma , float *gamma , float *ori_gamma , float *Cgamma , float *ori_Cgamma , float *grad_beta , float * ori_grad_beta , float *grad_sigma , float *ori_grad_sigma , float *grad_gamma , float *ori_grad_gamma , float *rho, float *ori_rho, float *grad_rho, float *ori_grad_rho, float *CSgamma, float *ori_CSgamma)
{
    // Replace the proposed values with the original
    likelihood[1] = likelihood[0];
    
    Replace<<<1,K_star>>>(beta,ori_beta);
    Replace<<<1,K>>>(rho,ori_rho);
    Replace<<<1,K>>>(sigma,ori_sigma);
    
    Replace<<<K*V/NT,NT>>>(gamma,ori_gamma);
    Replace<<<K*V/NT,NT>>>(Cgamma,ori_Cgamma);
    Replace<<<K*V/NT,NT>>>(CSgamma,ori_CSgamma);
    
    Replace<<<1,K_star>>>(grad_beta,ori_grad_beta);
    Replace<<<1,K>>>(grad_sigma,ori_grad_sigma);
    Replace<<<K*V/NT,NT>>>(grad_gamma,ori_grad_gamma);
    Replace<<<1,K>>>(grad_rho,ori_grad_rho);
    
    
    hipDeviceSynchronize();
}



// This function generates the momentum. CUDA won't generate odd number of variables hence the if statement. 
void GENERATE_MOMENTUM(float *mom_beta , float *mom_sigma , float *mom_gamma,hiprandGenerator_t gen , float *mom_beta_even , float *mom_sigma_even , float *mom_gamma_even , float *Mass_sigma , float *Mass_beta, float *mom_rho, float *mom_rho_even, float *Mass_rho)
{
    // Generate the N(0,1) momentum for the variables of interest
    // Overall means
    if ( (HK_star%2)==0 ) {
        hiprandGenerateNormal(gen , mom_beta , K_star , 0.0f , 1.0f);
    } else {
        hiprandGenerateNormal(gen, mom_beta_even,  K_star+1 , 0.0f, 1.0f);
        Replace<<<1,HK_star>>>(mom_beta_even,mom_beta);
    }
    // Marginal standard deviations
    if ( (HK%2)==0 ) {
        hiprandGenerateNormal(gen, mom_sigma, HK   , 0.0f, 1.0f);
        hiprandGenerateNormal(gen, mom_rho,  HK   , 0.0f, 1.0f);
    } else {
        hiprandGenerateNormal(gen, mom_sigma_even,  HK+1 , 0.0f, 1.0f);
        Replace<<<1,HK>>>(mom_sigma_even,mom_sigma);
        hiprandGenerateNormal(gen, mom_rho_even,    HK+1 , 0.0f, 1.0f);
        Replace<<<1,HK>>>(mom_rho_even,mom_rho);
    }
    // gamma vactors
    hiprandGenerateNormal(gen, mom_gamma, HK*V    , 0.0f, 1.0f);
    hipDeviceSynchronize();

    // Now scale the scalars by their masses
    CrossVector<<<1,HK_star>>>(mom_beta,Mass_beta);
    CrossVector<<<1,HK>>>(mom_sigma,Mass_sigma);
    CrossVector<<<1,HK>>>(mom_rho,  Mass_rho);
    hipDeviceSynchronize();
}





// Finds the kinetic energy
void FIND_KINETIC(float *mom_beta , float *mom_sigma , float *mom_gamma , double *Kinetic , double *kin_part , double *Hkin_part , float *Hmom_beta , float *Hmom_sigma , float *HMass_sigma , float *HMass_beta , float *mom_rho , float *Hmom_rho , float *HMass_rho)
{
    // Execute the kernels and transfer things to the CPU
    KineticFirst<<<NB,NT>>>(mom_gamma,kin_part);
    hipDeviceSynchronize();
    hipMemcpy(Hkin_part,kin_part,NB*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(Hmom_beta,mom_beta,HK_star*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(Hmom_sigma, mom_sigma, HK*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(Hmom_rho  , mom_rho  , HK*sizeof(float),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Define the variables that will be used
    int s;
    double sum = 0;

    // Add the Kinetic parts together
    for (s=0 ; s<NB ; s++) {
        sum += Hkin_part[s]; 
    }
    for (s=0 ; s<HK_star ; s++){
        sum += (Hmom_beta[s]*Hmom_beta[s])/HMass_beta[s];
    }

    for (s=0 ; s<HK ; s++){
        sum += (Hmom_sigma[s]*Hmom_sigma[s])/HMass_sigma[s]; 
    }

    for (s=0 ; s<HK ; s++){
        sum += (Hmom_rho[s]*Hmom_rho[s])/HMass_rho[s]; 
    }

    // Multiply by half
    Kinetic[0] = 0.5*sum;

}




// Updates the momentum vector according to HMC
void UPDATE_MOMENTUM(float size , float *mom_beta , float *grad_beta , float *mom_sigma , float *grad_sigma , float *mom_gamma , float * grad_gamma , float *mom_rho , float *grad_rho , float *Sign_rho)
{

    Update<<<1,HK_star>>>(mom_beta,grad_beta,size);
    Update<<<1,HK>>>(mom_sigma, grad_sigma, size);
    UpdateSecond<<<1,HK>>>(mom_rho,grad_rho,size,Sign_rho);
    Update<<<HK*V/NT,NT>>>(mom_gamma,grad_gamma,size);
    hipDeviceSynchronize();

}

// Updates the parameter vector according to HMC
void UPDATE_PARAMS(float size , float *beta , float *mom_beta , float *sigma , float *mom_sigma , float *gamma , float *mom_gamma , float *Mass_sigma , float *Mass_beta , float *rho, float *Hrho , float *mom_rho ,  float *Mass_rho , float *Sign_rho)
{
    // Update the parameters according to the Leapfrog scheme. Remember the Mass vectors contain standard deviations
    UpdateScalars<<<1,HK_star>>>(beta,mom_beta,size,Mass_beta);
    UpdateScalars<<<1,HK>>>(sigma,mom_sigma,size,Mass_sigma);
    UpdateScalars<<<1,HK>>>(rho,mom_rho,size,Mass_rho);
    Update<<<HK*V/NT,NT>>>(gamma,mom_gamma,size);
    hipDeviceSynchronize();

    // Bring the correlation parameters back to see if the boundary conditions are satisfied
    hipMemcpy(Hrho , rho , HK*sizeof(float) , hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    int s;
    float tmp;

    for (s=0 ; s<HK ; s++) {
        // Upper bounds
        if (Hrho[s] > RHO_UPPER) {
            tmp = RHO_UPPER - (Hrho[s]-RHO_UPPER);
            SetElement<<<1,1>>>(rho,s,tmp);
            SetElement<<<1,1>>>(Sign_rho , s , -1.0f);
        }
        // Lower bounds
        if (Hrho[s] < RHO_LOWER) {
            tmp = RHO_LOWER + (RHO_LOWER-Hrho[s]);
            SetElement<<<1,1>>>(rho,s,tmp);
            SetElement<<<1,1>>>(Sign_rho,s,-1.0f);
        }
    }
    hipDeviceSynchronize();

}





// For when a move is rejected
void REVERT_STATE(float *beta , float *ori_beta , float *sigma , float *ori_sigma , float *gamma , float *ori_gamma , float *Cgamma , float *ori_Cgamma , float *grad_beta , float * ori_grad_beta , float *grad_sigma , float *ori_grad_sigma , float *grad_gamma , float *ori_grad_gamma, float *rho, float *ori_rho, float *grad_rho, float *ori_grad_rho, float *CSgamma, float *ori_CSgamma)
{

Replace<<<1,K_star>>>(ori_beta,beta);
Replace<<<1,K>>>(ori_sigma,sigma);
Replace<<<1,K>>>(ori_rho,rho);

Replace<<<K*V/NT,NT>>>(ori_gamma,gamma);
Replace<<<K*V/NT,NT>>>(ori_Cgamma,Cgamma);
Replace<<<K*V/NT,NT>>>(ori_CSgamma,CSgamma);

Replace<<<1,K_star>>>(ori_grad_beta,grad_beta);
Replace<<<1,K>>>(ori_grad_sigma,grad_sigma);
Replace<<<1,K>>>(ori_grad_rho,grad_rho);
Replace<<<K*V/NT,NT>>>(ori_grad_gamma,grad_gamma);

hipDeviceSynchronize();

}





// Find the kinetic energy for the first part where only the gamma vectors are updated
void FIND_KINETIC_GAMMA(float *mom_gamma , double *Kinetic , double *kin_part , double *Hkin_part)
{
    // Execute the kernels and transfer to the CPU 
    KineticFirst<<<NB,NT>>>(mom_gamma,kin_part);
    hipDeviceSynchronize();
    hipMemcpy(Hkin_part,kin_part,NB*sizeof(double),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Define the variables that will be used
    int s=0;
    double sum=0;

    // Add things up
    for (s=0 ; s<NB ; s++) {
        sum += Hkin_part[s];
    }

    // Multiply by half
    Kinetic[0] = 0.5*sum;
}


// Saves a snapshot of the parameters
void SNAPSHOT(float *Hsigma , float *ori_sigma , float *Hrho , float *ori_rho , float *Hbeta , float *ori_beta , float *big , float *ori_gamma )
{   
    FILE *STARTING;
    int kk;

    /* Transfer the parameters to the CPU */
    hipMemcpy(Hsigma,ori_sigma,HK*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(Hrho  ,ori_rho,HK*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(Hbeta,ori_beta,HK_star*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(big,ori_gamma,HK*V_extended*sizeof(float),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    /* Write to the file */
    STARTING = fopen("./outputs/starting.txt","w");
    for (kk=0 ; kk<HK_star; kk++) {
        fprintf(STARTING,"%.10f\n",Hbeta[kk]);
    }
    for (kk=0 ; kk<HK ; kk++) {
        fprintf(STARTING,"%.10f\n",Hsigma[kk]);
    }
    for (kk=0 ; kk<HK ; kk++) {
        fprintf(STARTING,"%.10f\n",Hrho[kk]);
    }
    for (kk=0 ; kk<HK*V_extended ; kk++) {
        fprintf(STARTING,"%.10f\n",big[kk]);
    }
    fclose(STARTING);
}



/* Updates the study random effects */
void UPDATE_RFX(double *Hrfx , double *rfx, float *HZB, float *ZB, int *Hcounts, float *sigma, float *Cgamma, float *vol, float *COVARIATES, unsigned long *RNG, int* author)
{
    /* Find the sum over voxels and move to host */
    rfxSum<<<CUBLAS_TMP,512>>>(sigma , Cgamma , vol , COVARIATES , rfx );
    hipDeviceSynchronize();
    hipMemcpy(Hrfx,rfx,HI*sizeof(double),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    /* Add the constant terms */
    int i;
    hipMemcpy(HZB, ZB, HI*sizeof(float), hipMemcpyDeviceToHost);
    for (i=0 ; i<HI ; i++) {
        Hrfx[i] *= VOXEL_VOL*exp((double)HZB[i]);
    }

    /* Sample the random effect terms from their Gamma full conditionals */
    int n_authors = author[HI-1] + 1 ;
    int author_first, author_last, flag, j;
    double shape, rate, tmp;
    for (i=0 ; i<n_authors ; i++) {
        
        /* Find the first study from the i-th author */ 
        flag=0; j=-1;
        while (flag==0){
            j += 1;
            flag = ( i == (author[j]) );
        }
        author_first = j;
        /* Find the last study from the i-th author */ 
        for (j=author_first ; j<HI ; j++) {
            if (author[j] == i){
                author_last = j;
            }
        }
        /* Find the shape and the rate of the Gamma full conditional */
        shape = rfx_phi; rate = rfx_phi;
        for (j=author_first ; j<=author_last ; j++) {
            shape += (double)Hcounts[j];
            rate  += Hrfx[j];
        } 
        /* Draw the new random effect */
        tmp = rgamma(shape,rate,RNG);
        /* Save the draw */
        for (j=author_first ; j<=author_last ; j++) {
            Hrfx[j] = tmp;
        }
        /* Print something to make sure */
        if (i==100) {
            printf("\nStudy %d First %d Last %d Shape %.5f Rate %.5f",i,author_first,author_last,shape,rate);
        }
    }


    /* Copy the random effects to the GPU */ 
    hipMemcpy(rfx,Hrfx,HI*sizeof(double),hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    /* Copy the random effects in the  COVARIATES array */
    SaveRFX<<<CUBLAS_TMP,512>>>(rfx,COVARIATES);
    hipDeviceSynchronize();


}



